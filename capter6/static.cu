#include "hip/hip_runtime.h"
#include "error.cuh"


// 静态全局内存变量。  
__device__ int d_x = 1;
__device__ int d_y[2] = {2, 3};


// 核函数。
__global__ void add_array()
{
    d_y[0] += d_x;
    d_y[1] += d_x;

    printf("d_y: {%d, %d}\n", d_y[0], d_y[1]);
}
__global__ void add_var()
{
    d_x += 2;

    printf("d_x: %d\n", d_x);
}
__global__ void display()
{
    printf("d_x: %d, d_y: {%d, %d}\n", d_x, d_y[0], d_y[1]);
}


int main()
{
    display<<<1, 1>>>();
    add_array<<<1, 1>>>();
    add_var<<<1, 1>>>();
    CHECK(hipDeviceSynchronize());
    
    int h_y[2] = {10, 20};

    // hipError_t hipMemcpyToSymbol(HIP_SYMBOL(const void *symbol), const void *src, size_t count, 
    //     size_t offset=0, hipMemcpyKind kind=hipMemcpyHostToDevice);
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_y), h_y, sizeof(int)));
    display<<<1, 1>>>();

    // hipError_t hipMemcpyFromSymbol(void *dst, HIP_SYMBOL(const void *symbol), size_t count, 
    //     size_t offset=0, hipMemcpyKind kind=hipMemcpyDeviceToHost);
    CHECK(hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_y), sizeof(int) * 2));
    printf("host, d_y: %d, %d\n", h_y[0], h_y[1]);
    display<<<1, 1>>>();

    return 0;
}




