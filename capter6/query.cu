#include "common/error.cuh"
#include <stdlib.h>


int main(int argc, char *argv[])
{
    int device_id = 0;
    if (argc > 1) device_id = atoi(argv[1]);

    CHECK(hipSetDevice(device_id));

    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, device_id));

    printf("Device id: %d\n", device_id);
    printf("Device name: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Amount of global memory: %g GB\n", prop.totalGlobalMem/(1024.0*1024*1024));
    printf("Amount of constant memory: %g KB\n", prop.totalConstMem/1024.0);
    printf("Maximum grid size: %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Maximum block size: %d, %d, %d\n", prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
    printf("Number of SMs: %d\n", prop.multiProcessorCount);
    printf("Maximum amount of shared memory per block: %g KB\n", prop.sharedMemPerBlock/1024.0);
    printf("Maximum amount of shared memory per SM: %g KB\n", prop.sharedMemPerMultiprocessor/1024.0);
    printf("Maximum number of registers per block: %d K\n", prop.regsPerBlock/1024);
    printf("Maximum number of registers per SM: %d K\n", prop.regsPerMultiprocessor/1024);
    printf("Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Maximum number of threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);

    return 0;
}
