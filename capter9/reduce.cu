#include "hip/hip_runtime.h"
#include "../common/error.cuh"
#include "../common/floats.hpp"
#include "../common/clock.cuh"


__global__ void reduce(real *x, real *y, const int N)
{
    int tid = threadIdx.x;
    int ind = tid + blockIdx.x * blockDim.x;

    extern __shared__ real curr_x[];
    curr_x[tid] = (ind < N) ? x[ind] : 0.0;    

    for (int offset = blockDim.x/2 ; offset > 0 ; offset /= 2)
    {
        if (tid < offset)
        {
            curr_x[tid] += curr_x[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        y[blockIdx.x] = curr_x[0];
    }
}

__global__ void reduce2(real *x, real *y, const int N)
{
    int tid = threadIdx.x;
    int ind = tid + blockIdx.x * blockDim.x;

    extern __shared__ real curr_x[];
    curr_x[tid] = (ind < N) ? x[ind] : 0.0;    

    for (int offset = blockDim.x/2 ; offset > 0 ; offset /= 2)
    {
        if (tid < offset)
        {
            curr_x[tid] += curr_x[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        // 原子函数 atomicAdd(*address, val).
        atomicAdd(y, curr_x[0]);
    }
}



int main()
{
    int N = 1e8;
    int M = N * sizeof(real);

    int bSize = 32;
    int gSize = (N + bSize - 1)/bSize;

    cout << FLOAT_PREC << endl;

    real *h_x, *h_y;
    h_x = new real[N];
    h_y = new real[gSize];
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = 1.23;
    }

    cudaClockStart

    real *d_x, *d_y;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMalloc(&d_y, gSize*sizeof(real)));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyDefault));

    cudaClockCurr

    reduce<<<gSize, bSize, (bSize+1)*sizeof(real)>>>(d_x, d_y, N);

    CHECK(hipMemcpy(h_y, d_y, gSize*sizeof(real), hipMemcpyDefault));
    real res = 0;
    for(int i = 0; i < gSize; ++i)
    {
        res += h_y[i];
    }
    cout << "reduce result: " << res << endl;

    cudaClockCurr

    reduce<<<gSize, bSize, (bSize)*sizeof(real)>>>(d_x, d_y, N);
    
    CHECK(hipMemcpy(h_y, d_y, gSize*sizeof(real), hipMemcpyDefault));
    res = 0.0;
    for(int i = 0; i < gSize; ++i)
    {
        res += h_y[i];
    }
    cout << "reduce result: " << res << endl;

    cudaClockCurr
    
    real *d_y2, *h_y2;
    h_y2 = new real(0.0);
    CHECK(hipMalloc(&d_y2, sizeof(real)));

    // 采用原子函数、共享内存的核函数归约，
    // 由于减少了主机和设备间的数据传输，效率得以提高。
    reduce2<<<gSize, bSize, (bSize)*sizeof(real)>>>(d_x, d_y2, N);

    CHECK(hipMemcpy(h_y2, d_y2, sizeof(real), hipMemcpyDefault));
    cout << "reduce2 result: " << *h_y2 << endl;

    cudaClockCurr

    delete[] h_x;
    delete[] h_y;
    delete h_y2;
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_y2));

    return 0;
}







