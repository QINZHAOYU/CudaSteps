#include "hip/hip_runtime.h"
#include "../common/error.cuh"
#include "../common/floats.hpp"
#include <iomanip>
#include <string>
#include <fstream>

#define TILE_DIM  32

__constant__ int c_TILE_DIM = 32;  // 设备内存中线程块中矩阵维度（线程块大小，最大1024）。

void show(const real *matrix, const int N, std::string outfile, std::string title);
__global__ void transpose1(const real *src, real *dst, const int N);
__global__ void transpose2(const real *src, real *dst, const int N);
__global__ void transpose3(const real *src, real *dst, const int N);
__global__ void transpose4(const real *src, real *dst, const int N);



int main()
{
    // 由于显存 2 GB，float 为 4 字节，double 为 8 字节，所以在 transpose3, transpose4中：
    // float 矩阵维度不能超过 726；
    // double 矩阵维度不能超过 512；
    const int N = 500;  
    const int M = N * N * sizeof(real);

    int SIZE = 0;
    CHECK(hipMemcpyFromSymbol(&SIZE, HIP_SYMBOL(c_TILE_DIM), sizeof(int)));   

    const int grid_size_x = (N + SIZE - 1)/SIZE; // 获取网格大小。
    const int grid_size_y = grid_size_x;

    const dim3 block_size(SIZE, SIZE);
    const dim3 grid_size(grid_size_x, grid_size_y);

    real *h_matrix_org, *h_matrix_res;
    h_matrix_org = new real[N*N];
    h_matrix_res = new real[N*N];
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            h_matrix_org[i * N + j] = i*1.0e-2;
        }   
    }
    // show(h_matrix_org, N, "result.txt", "origin matrix");
    
    real *d_matrix_org, *d_matrix_res;
    CHECK(hipMalloc(&d_matrix_org, M));
    CHECK(hipMalloc(&d_matrix_res, M));
    CHECK(hipMemcpy(d_matrix_org, h_matrix_org, M, hipMemcpyDefault));    

    float elapsed_time = 0;
    float curr_time = 0;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    // 矩阵转置（全局内存合并读取、非合并写入）。
    transpose1<<<grid_size, block_size>>>(d_matrix_org, d_matrix_res, N);     
    CHECK(hipMemcpy(h_matrix_res, d_matrix_res, M, hipMemcpyDefault));
    // show(h_matrix_res, N, "result.txt", "transpose1");

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&curr_time, start, stop));    
    printf("matrix transpose1 time cost: %f ms.\n", curr_time - elapsed_time);
    elapsed_time = curr_time;

    // 矩阵转置（全局内存非合并读取、合并写入）。
    transpose2<<<grid_size, block_size>>>(d_matrix_org, d_matrix_res, N);     
    CHECK(hipMemcpy(h_matrix_res, d_matrix_res, M, hipMemcpyDefault));
    // show(h_matrix_res, N, "matrix.txt", "transpose2");

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&curr_time, start, stop));    
    printf("matrix transpose2 time cost: %f ms.\n", curr_time - elapsed_time);
    elapsed_time = curr_time;

    // 矩阵转置（通过共享内存全局内存合并读写）。
    transpose3<<<grid_size, block_size>>>(d_matrix_org, d_matrix_res, N);     
    CHECK(hipMemcpy(h_matrix_res, d_matrix_res, M, hipMemcpyDefault));
    // show(h_matrix_res, N, "result.txt", "transpose3");

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&curr_time, start, stop));    
    printf("matrix transpose3 time cost: %f ms.\n", curr_time - elapsed_time);
    elapsed_time = curr_time;

    // 矩阵转置（通过共享内存、bank处理，实现全局内存合并读写）。
    transpose4<<<grid_size, block_size>>>(d_matrix_org, d_matrix_res, N);     
    CHECK(hipMemcpy(h_matrix_res, d_matrix_res, M, hipMemcpyDefault));
    // show(h_matrix_res, N, "result.txt", "transpose3");

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&curr_time, start, stop));    
    printf("matrix transpose4 time cost: %f ms.\n", curr_time - elapsed_time);
    elapsed_time = curr_time;    

    delete[] h_matrix_res;
    delete[] h_matrix_org;
    CHECK(hipFree(d_matrix_org));
    CHECK(hipFree(d_matrix_res));

    return 0;
}


void show(const real *x, const int N, std::string outfile, std::string title)
{
    std::fstream out(outfile, std::ios::app);
    if (!out.is_open())
    {
        std::cerr << "invalid output file: " << outfile << endl;
        return;
    }

    out << "\n\n----------------" << title << endl;

    for (int i = 0; i < N; ++i)
    {
        out << endl;
        for (int j = 0; j < N; ++j)
        {
            out << std::setw(6) << x[i * N + j];
        }
    }
}

__global__ void transpose1(const real *src, real *dst, const int N)
{
    const int nx = threadIdx.x + blockIdx.x * c_TILE_DIM;
    const int ny = threadIdx.y + blockIdx.y * c_TILE_DIM;

    if (nx < N && ny < N)
    {
        // 矩阵转置（合并读取、非合并写入）。
        dst[nx*N + ny] = src[ny*N + nx];
    }
}

__global__ void transpose2(const real *src, real *dst, const int N)
{
    const int nx = threadIdx.x + blockIdx.x * c_TILE_DIM;
    const int ny = threadIdx.y + blockIdx.y * c_TILE_DIM;

    if (nx < N && ny < N)
    {
        // 矩阵转置（非合并读取、合并写入）。
        dst[ny*N + nx] = __ldg(&src[nx*N + ny]);   // 显示调用 `__ldg()` 函数缓存全局内存。 
    }
}

__global__ void transpose3(const real *src, real *dst, const int N)
{
    // 正常的做法中，全局内存的读写必有一个是非合并访问。
    // 现在通过将非合并访问转移到共享内存，利用共享内存的高性能（100倍全局内存），提高计算速度：  
    // 1. 首先将全局内存拷贝到线程块的共享内存；
    // 2. 然后从共享内存非合并访问，读取数据，合并写入全局内存。

    __shared__ real s_mat[TILE_DIM][TILE_DIM];  //二维静态共享内存，存储线程块内的一片矩阵。

    int bx = blockIdx.x * blockDim.x;  // 当前线程块首线程在网格中列索引。
    int by = blockIdx.y * blockDim.y;  // 当前线程块首线程在网格中行索引。

    int tx = threadIdx.x + bx;  // 当前线程在网格中列索引。
    int ty = threadIdx.y + by;  // 当前线程在网格中行索引。

    if (tx < N && ty < N)
    {
        // 全局内存合并访问，共享内存合并访问。
        s_mat[threadIdx.y][threadIdx.x] = src[ty * N + tx]; // 全局内存中二维矩阵一维存储。
    }
    __syncthreads();
    
    // 全局内存合并访问。
    if (tx < N && ty < N)
    {
        // 局部矩阵转置和全局内存合并写入。
        int x = by + threadIdx.x;
        int y = bx + threadIdx.y;
        dst[y * N + x] = s_mat[threadIdx.x][threadIdx.y]; 
    }
}

__global__ void transpose4(const real *src, real *dst, const int N)
{
    // 通过修改数组行大小，错开数组元素在共享内存bank中的分布，
    // 避免线程束的 32路bank冲突。
    __shared__ real s_mat[TILE_DIM][TILE_DIM + 1];  

    int bx = blockIdx.x * blockDim.x; 
    int by = blockIdx.y * blockDim.y; 

    int tx = threadIdx.x + bx;  
    int ty = threadIdx.y + by;  

    if (tx < N && ty < N)
    {
        s_mat[threadIdx.y][threadIdx.x] = src[ty * N + tx]; 
    }
    __syncthreads();
    
    if (tx < N && ty < N)
    {
        int x = by + threadIdx.x;
        int y = bx + threadIdx.y;
        dst[y * N + x] = s_mat[threadIdx.x][threadIdx.y]; 
    }
}